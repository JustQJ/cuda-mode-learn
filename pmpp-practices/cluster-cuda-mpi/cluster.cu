#include "hip/hip_runtime.h"

#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<mpi.h>



void random_data(float *data, uint num_points){
    // for(uint i=0; i<num_points; i++){
    //     data[i] = (float)rand()/(float)RAND_MAX;
    // }
}

void store_output(float *data, int dimx, int dimy, int dimz){
    //store the output data
}


void server_process(int dimx, int dimy, int dimz){
    int np;
    MPI_Comm_size(MPI_COMM_WORLD, &np); // get the number of processes

    uint num_comp_nodes = np-1, first_node=0, last_node=np-2; //compute nodes, first and last compute node id
    uint num_points = dimx*dimy*dimz; //number of data points
    uint num_bytes = num_points*sizeof(float); //number of bytes to store the data

    float *input=0, *output=0;
    input = (float*)malloc(num_bytes);
    output = (float*)malloc(num_bytes);
    if(input==NULL || output==NULL){
        printf("Error: unable to allocate memory\n");
        MPI_Abort(MPI_COMM_WORLD, 1);
       
    }

    //initialize input data
    random_data(input, num_points);

    // compute the number of points to be processed by each compute node, first and last node may have different number of points 
    int edge_num_points = dimx*dimy*(dimz/num_comp_nodes + 4); //4 is needed for the halo from the previous or next node, first and last node
    int int_num_points = dimx*dimy*(dimz/num_comp_nodes + 8); //8 is needed for the halo from the previous and next node, internal nodes

    float *send_address = input;

    MPI_Send(send_address, edge_num_points, MPI_FLOAT, first_node, 0, MPI_COMM_WORLD); //send data to the first node
    send_address += dimx*dimy*(dimz/num_comp_nodes - 4); //move the pointer to the next node, because we need to send the halo, so -4
    for(int process=1; process<last_node; process++){
        MPI_Send(send_address, int_num_points, MPI_FLOAT, process, 0, MPI_COMM_WORLD); //send data to the internal nodes
        send_address += dimx*dimy*(dimz/num_comp_nodes); //move the pointer to the next node
    }
    MPI_Send(send_address, edge_num_points, MPI_FLOAT, last_node, 0, MPI_COMM_WORLD); //send data to the last node


    //receive the results

    MPI_Barrier(MPI_COMM_WORLD); //wait for all the processes to finish
    MPI_Status status;
    for(int process=0; process<num_comp_nodes; process++){
        MPI_Recv(output+process*num_points/num_comp_nodes, num_points/num_comp_nodes, MPI_FLOAT, process, 0, MPI_COMM_WORLD, &status);
        //process the results
    }
    store_output(output, dimx, dimy, dimz);
    free(input);
    free(output);




}


__global__ void stencil_kernel(float *output, float *input, int dimx, int dimy, int dimz){
    // idx for three dimensions
    int x = threadIdx.x + blockIdx.x*blockDim.x;    
    int y = threadIdx.y + blockIdx.y*blockDim.y;
    int z = threadIdx.z + blockIdx.z*blockDim.z;

    if(x<dimx && y<dimy && z<dimz){
        int idx = x + y*dimx + z*dimx*dimy + 4*dimx*dimy; //4 is needed for the halo from the previous and next node
        output[idx] = 0.25f*(input[idx]+input[idx]+input[idx]+input[idx]+input[idx]);  // is not right, just a case  + 0.125f*(input[idx+1]+ input[idx-1] + input[idx+dimx] + input[idx-dimx] + input[idx+dimx*dimy] + input[idx-dimx*dimy]);
    }

}


void call_stencil_kernel(float *output, float *input, int dimx, int dimy, int dimz, hipStream_t stream){
    dim3 block(8, 8, 8);
    dim3 grid((dimx+block.x-1)/block.x, (dimy+block.y-1)/block.y, (dimz-8+block.z-1)/block.z);  //remove the halo, 4 left, 4 right
    
    stencil_kernel<<<grid, block, 0, stream>>>(output, input, dimx, dimy, dimz-8);
    
}

void compute_process(int dimx, int dimy, int dimz, int nreps){
    int np, pid;
    MPI_Comm_size(MPI_COMM_WORLD, &np); // get the number of processes
    MPI_Comm_rank(MPI_COMM_WORLD, &pid); // get the process id

    int server_process = np-1; //server process id
    uint num_points = dimx*dimy*(dimz+8); //number of data points, 8 is needed for the halo from the previous and next node
    uint num_bytes = num_points*sizeof(float); //number of bytes to store the data
    uint num_halo_points = dimx*dimy*4; //number of halo points
    uint num_halo_bytes = num_halo_points*sizeof(float); //number of bytes to store the halo data

    //allocate memory for the data
    float *h_input = (float*)malloc(num_bytes);
    //set zero to the input data
    memset(h_input, 0, num_bytes);
    float *d_input = NULL;
    hipMalloc(&d_input, num_bytes);
    float *rcv_address = h_input + ((pid==0)?num_halo_points:0); //move the pointer to the correct position, first process has no previous halo
    MPI_Recv(rcv_address, num_points, MPI_FLOAT, server_process, MPI_ANY_TAG, MPI_COMM_WORLD, MPI_STATUS_IGNORE); //receive the data
    hipMemcpy(d_input, h_input, num_bytes, hipMemcpyHostToDevice); //copy the data to the device

    //allocation  output
    float *h_output = (float*)malloc(num_bytes);
    float *d_output = NULL;
    hipMalloc(&d_output, num_bytes);

    float *h_left_boundary = NULL, *h_right_boundary = NULL;  //for send data to the previous and next node
    float *h_left_halo = NULL, *h_right_halo = NULL; //for receive data from the previous and next node

    //pinned memory for the asynchronous data transfer, if not the pinned memory, cudaMemcpuAsync will be not turlly asynchronous
    hipHostAlloc(&h_left_boundary, num_halo_bytes, hipHostMallocDefault);
    hipHostAlloc(&h_right_boundary, num_halo_bytes, hipHostMallocDefault);
    hipHostAlloc(&h_left_halo, num_halo_bytes, hipHostMallocDefault);
    hipHostAlloc(&h_right_halo, num_halo_bytes, hipHostMallocDefault);

    //create streams, for the computation and data transfer
    hipStream_t stream0, stream1;
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);


    int left_neighbor = (pid==0)?MPI_PROC_NULL:pid-1; //left neighbor
    int right_neighbor = (pid==np-2)?MPI_PROC_NULL:pid+1; //right neighbor

    //do computing iterations

    int left_halo_offset = 0, right_halo_offset = dimx*dimy*(dimz+4); //offset for the halo data
    int left_stage1_offset = 0;
    int right_stage1_offset = dimx*dimy*(dimz-4);
    int stage2_offset = num_halo_points;

    MPI_Barrier(MPI_COMM_WORLD); //wait for all the processes to start
    double start_time = MPI_Wtime();
    for(int i=0; i<nreps; i++){
        //compute the left halo and right halo first, need to send them to the previous and next node
        call_stencil_kernel(d_output+left_stage1_offset, d_input+left_stage1_offset, dimx, dimy, 12, stream0);
        call_stencil_kernel(d_output+right_stage1_offset, d_input+right_stage1_offset, dimx, dimy, 12, stream0);

        //compute the internal points
        call_stencil_kernel(d_output+stage2_offset, d_input+stage2_offset, dimx, dimy, dimz, stream1);

        //copy the halo data
        hipMemcpyAsync(h_left_boundary, d_output+num_halo_points, num_halo_bytes, hipMemcpyDeviceToHost, stream0);
        hipMemcpyAsync(h_right_boundary, d_output+right_stage1_offset+num_halo_points, num_halo_bytes, hipMemcpyDeviceToHost, stream0);
        hipStreamSynchronize(stream0);
        // hipMemcpy(h_left_boundary, d_output+num_halo_points, num_halo_bytes, hipMemcpyDeviceToHost);
        // hipMemcpy(h_right_boundary, d_output+right_stage1_offset+num_halo_points, num_halo_bytes, hipMemcpyDeviceToHost);


        //send data to the previous and next node and receive the halo data

        MPI_Sendrecv(h_left_boundary, num_halo_points, MPI_FLOAT, left_neighbor, i, h_right_halo, num_halo_points, MPI_FLOAT, right_neighbor, i, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
        MPI_Sendrecv(h_right_boundary, num_halo_points, MPI_FLOAT, right_neighbor, i, h_left_halo, num_halo_points, MPI_FLOAT, left_neighbor, i, MPI_COMM_WORLD, MPI_STATUS_IGNORE);

        //copy the halo data to the device
        hipMemcpyAsync(d_output+left_halo_offset, h_left_halo, num_halo_bytes, hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(d_output+right_halo_offset, h_right_halo, num_halo_bytes, hipMemcpyHostToDevice, stream0);

        hipDeviceSynchronize(); //wait for all tasks to finish

        //swap the input and output
        float *temp = d_input;
        d_input = d_output;
        d_output = temp;
    }

    double end_time = MPI_Wtime();
    printf("Process %d, elapsed time: %f seconds\n", pid, end_time-start_time);

    //wait all process
    
    //undo the swap in last iteration
    float *temp = d_input;
    d_input = d_output;
    d_output = temp;

    //send data to the server, skip all halo data
    hipMemcpy(h_output, d_output, num_bytes, hipMemcpyDeviceToHost);
    float *send_address = h_output + ((pid==0)?num_halo_points:0);
    MPI_Send(send_address, dimx*dimy*dimz, MPI_FLOAT, server_process, 0, MPI_COMM_WORLD);
    // MPI_Barrier(MPI_COMM_WORLD);
    //free

    free(h_input);
    free(h_output);
    hipFree(d_input);
    hipFree(d_output);
    hipHostFree(h_left_boundary);
    hipHostFree(h_right_boundary);
    hipHostFree(h_left_halo);
    hipHostFree(h_right_halo);
    hipStreamDestroy(stream0);
    hipStreamDestroy(stream1);



}


/*
nvcc -o cluster cluster.cu  -I/usr/lib/x86_64-linux-gnu/openmpi/include -L/usr/lib/x86_64-linux-gnu/openmpi/lib -lmpi -lcudart
mpirun -np 4 ./cluster
*/



// int main(){

//     int num_bytes = 32*32*32*sizeof(float);

//     float *d_input = NULL;
//     hipMalloc(&d_input, num_bytes);

//     //allocation  output
//     float *d_output = NULL;
//     hipMalloc(&d_output, num_bytes);
//     hipStream_t stream0, stream1;
//     hipStreamCreate(&stream0);
//     hipStreamCreate(&stream1);
//     call_stencil_kernel(d_output, d_input, 32, 32, 32, stream0);

//     return 0;

// }


int main(int argc, char* argv[]){
    int pad=0, dimx=2*480+pad, dimy=2*480, dimz=2*400, nreps=100;

    int pid=-1, np=-1;

    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &pid); // get the process id
    MPI_Comm_size(MPI_COMM_WORLD, &np); // get the number of processes

    if(np<3){
        printf("This program requires at least 3 processes\n");
        MPI_Abort(MPI_COMM_WORLD, 1);
        MPI_Finalize();
        return 1;
    }
    
    //record the start time
    double start_time = MPI_Wtime();


    if(pid<np-1){
        compute_process(dimx, dimy, dimz/(np-1), nreps); //compute 
    }else{
        server_process(dimx, dimy, dimz); //generate data, assign work and collect results
    }

    //record the end time
    double end_time = MPI_Wtime();
    if(pid==np-1){
        printf("Elapsed time: %f seconds\n", end_time-start_time);
    }

    MPI_Finalize();
    return 0;
}